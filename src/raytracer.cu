#include "hip/hip_runtime.h"
#include "raytracer.h"

#define QSIZE 5;
#define BLOCK_SIZE 8

typedef struct QItem {
    __device__ QItem() {}
    __device__ QItem(const Ray& r, const float3& m, int d) : ray(r), multiplier(m), depth(d) {}
    Ray ray;
    float3 multiplier;
    int depth;
} QItem;

typedef struct RayQ {
    __device__ RayQ() : start(0), end(0) {}
    __device__ void push(const QItem& item) {
        Q[end] = item;
        end = (end + 1) % QSIZE;
    }

    __device__ bool pop(QItem& item) {
        if (start == end)
            return false;
        item = Q[start];
        start = (start + 1) % QSIZE;
        return true;
    }

    int start;
    int end;
    QItem Q[5];
} RayQ;

__device__
int intersection(const RTScene& scene, const Ray& ray, float& t, int& type, int& meshNum, float& u, float& v, int* localStack) {
    float minT = 1e30f;
    int index = -1;
    type = 0;
    for (int i = 0; i < scene.numSpheres; ++i) {
        if (raySphereTest(ray, scene.spheres[i], t)) {
            if (t < minT) {
                index = i;
                minT = t;
            }
        }
    }

    /*
    float uu, vv;
    for (int m = 0; m < scene.numMeshes; ++m) {
        CudaMesh& mesh = scene.meshes[m];
        for (int i = 0; i < mesh.numTriangles; ++i) {
            if (rayTriangleTest(ray, mesh, mesh.triangles[i], t, uu, vv)) {
                if (t < minT) {
                    meshNum = m; index = i; type = 1; minT = t; u = uu; v = vv;
                }
            }
        }
    }
    */

    float uu, vv;
    float3 invRayDir = 1.0f / ray.dir;
    int stack[64];
    for (int m = 0; m < scene.numMeshes; ++m) {
        CudaMesh& mesh = scene.meshes[m];
        BVH* bvh = mesh.bvh;
        int idx = 0;
        stack[idx++] = 0;

        while (idx) {
            int i = stack[--idx];
            // BVH node = bvh[i];
            BVH node;
            float4 f1  = tex1Dfetch<float4>(mesh.bvhTex, 2 * i + 0);
            float4 f2  = tex1Dfetch<float4>(mesh.bvhTex, 2 * i + 1);
            node.min   = make_float3(f1.x, f1.y, f1.z);
            node.max   = make_float3(f2.x, f2.y, f2.z);
            node.left  = *(int*) &f1.w;
            node.right = *(int*) &f2.w;
            
            if (!RayAABBTest2(ray.pos, invRayDir, node.min, node.max, minT))
                continue;

            // if not a leaf node
            if (!node.isLeaf()) {
                if (node.left)
                    stack[idx++] = node.left;
                if (node.right)
                    stack[idx++] = node.right;
            } else { // if leaf
                Triangle leftTri = mesh.triangles[-node.left];
                if (rayTriangleTest2(ray, mesh, leftTri, t, uu, vv)) {
                    if (t < minT) {
                        meshNum = m; index = -node.left; type = 1; minT = t; u = uu; v = vv;
                    }
                }
                if (node.right < 0) {
                    Triangle rightTri = mesh.triangles[-node.right];
                    if (rayTriangleTest2(ray, mesh, rightTri, t, uu, vv)) {
                        if (t < minT) {
                            meshNum = m; index = -node.right; type = 1; minT = t; u = uu; v = vv;
                        }
                    }
                }
            }
        }
    }

    t = minT;
    return index;
}

__device__ float3 computeLighting(const RTScene& scene, const RTMaterial& mat, const float3& P, const float3& N, const float3& V, float t) {
    float3 color = make_float3(0, 0, 0);

    for (int i = 0; i < scene.numDirectionalLights; ++i) {
        float3 l = scene.lights[2 * i];
        float3 lightColor = scene.lights[2 * i + 1];

        color += lightColor * mat.kd * fmaxf(0.0f, dot(N, -l));
        color += lightColor * mat.ks * powf(fmaxf(0.0f, dot(V, reflect(l, N))), mat.power);
    }

    return color;
}

__device__ float3 traceRay(RayQ& Q, const QItem& item, const RTScene& scene, int* localStack) {
    if (item.depth >= 5)
        return make_float3(0, 0, 0);

    float t;
    const Ray& ray = item.ray;
    int type, meshNum;
    float u, v;
    int index = intersection(scene, ray, t, type, meshNum, u, v, localStack);
    if (index == -1)
        return make_float3(0, 0, 0);

    float3 color = make_float3(0, 0, 0);
    float3 p = ray.eval(t);
    float3 n;
    unsigned short matID;

    if (type == 0) { // sphere
        const Sphere& s = scene.spheres[index];
        n = normalize(p - s.pos);
        matID = s.matID;
    } else { // triangle
        n = scene.meshes[meshNum].getNormal(index, u, v);
        // if (dot(n, ray.dir) > 0)
        //     n = -n;
        matID = scene.meshes[meshNum].matID;
    }
    const RTMaterial& mat = scene.materials[matID];

    color += item.multiplier * computeLighting(scene, mat, p, n, -ray.dir, t);

    float3 reflectMult = item.multiplier * mat.ks;
    if (dot(reflectMult, reflectMult) < 0.1f)
        return color;

    float3 reflectDir = reflect(ray.dir, n);
    Ray reflectRay(p + 0.001f * reflectDir, reflectDir);
    QItem reflectItem(reflectRay, reflectMult, item.depth + 1);
    Q.push(reflectItem);

    return color;
}

__global__
void rayTraceKernel(hipSurfaceObject_t surface, int SW, int SH,
        float3 P, float3 UL, float3 DX, float3 DY, RTScene scene) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    //
    // __shared__ int stacks[32*BLOCK_SIZE*BLOCK_SIZE];
    // int* localStack = &stacks[32 * (BLOCK_SIZE*threadIdx.y + threadIdx.x)];
    int* localStack = NULL;

    if (x >= SW && y >= SH)
        return;

    RayQ Q;

    float3 pos2 = UL + x * DX + y * DY;
    QItem item(Ray(P, normalize(pos2 - P)), make_float3(1, 1, 1), 0);
    Q.push(item);

    float3 color = make_float3(0, 0, 0);

    while (Q.pop(item)) {
        color += traceRay(Q, item, scene, localStack);
    }

    uchar4 pixel = toPixel(color);
    surf2Dwrite(pixel, surface, x * sizeof(uchar4), y);
}

void RayTracer::Init(Scene& pgScene) {
    copyShader_ = PG::Shader(PG_RESOURCE_DIR "shaders/copy.vert", PG_RESOURCE_DIR "shaders/copy.frag");

    int SW = PG::Window::getWindowSize().x;
    int SH = PG::Window::getWindowSize().y;

    float quadVerts[] = {
        -1, 1,
        -1, -1,
        1, -1,

        -1, 1,
        1, -1,
        1, 1
    };
    glGenVertexArrays(1, &quadVAO_);
    glBindVertexArray(quadVAO_);
    glGenBuffers(1, &quadVBO_);
    glBindBuffer(GL_ARRAY_BUFFER, quadVBO_);
    glBufferData(GL_ARRAY_BUFFER, sizeof(quadVerts), quadVerts, GL_STATIC_DRAW);
    glEnableVertexAttribArray(copyShader_["vertex"]);
    glVertexAttribPointer(copyShader_["vertex"], 2, GL_FLOAT, GL_FALSE, 0, 0);

    glGenTextures(1, &glTexture_);
    glBindTexture(GL_TEXTURE_2D, glTexture_);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, SW, SH, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    check(hipGraphicsGLRegisterImage(&cudaTex_, glTexture_, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    scene = createRTSceneFromPGScene(pgScene);
}

void RayTracer::Free() {
    glDeleteBuffers(1, &quadVBO_);
    glDeleteVertexArrays(1, &quadVAO_);
    glDeleteTextures(1, &glTexture_);
}

void RayTracer::Render(Camera* camera) {
    int SW = PG::Window::getWindowSize().x;
    int SH = PG::Window::getWindowSize().y;

    // glBindTexture(GL_TEXTURE_2D, glTexture_);
    check(hipGraphicsMapResources(1, &cudaTex_));

    hipArray_t texture_ptr;
    check(hipGraphicsSubResourceGetMappedArray(&texture_ptr, cudaTex_, 0, 0))

    struct hipResourceDesc description;
    memset(&description, 0, sizeof(description));
    description.resType = hipResourceTypeArray;
    description.res.array.array = texture_ptr;

    hipSurfaceObject_t surf;
    check(hipCreateSurfaceObject(&surf, &description));

    auto p = camera->transform.position;
    auto dir = camera->GetForwardDir();
    auto up = camera->GetUpDir();
    auto right = camera->GetRightDir();
    float fov = camera->GetFOV();

    float d = SH / (2.0f * tan(fov));
    glm::vec3 ul = p + d * dir + up * (SH / 2.0f) - (SW / 2.0f) * right;

    float3 P, UL, DX, DY;
    P = make_float3(p.x, p.y, p.z);
    DX = make_float3(right.x, right.y, right.z);
    DY = make_float3(-up.x, -up.y, -up.z);
    UL = make_float3(ul.x, ul.y, ul.z);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim;
    gridDim.x = SW / blockDim.x + ((SW % blockDim.x) != 0);
    gridDim.y = SH / blockDim.y + ((SH % blockDim.y) != 0);
    gridDim.z = 1; 

    // std::cout << "num spheres: " << scene.numSpheres << std::endl;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("kernel Error: %s\n", hipGetErrorString(err));
    }
    rayTraceKernel<<<gridDim, blockDim>>>(surf, SW, SH, P, UL, DX, DY, scene);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("kernel Error: %s\n", hipGetErrorString(err));
    }

    check(hipDestroySurfaceObject(surf));
    check(hipGraphicsUnmapResources(1, &cudaTex_));
    // hipStreamSynchronize(0);
    check(hipDeviceSynchronize());


    PG::graphics::SetClearColor(1, 1, 1, 1);
    PG::graphics::Clear();
    copyShader_.Enable();
    glBindVertexArray(quadVAO_);
    graphics::Bind2DTexture(glTexture_, copyShader_["tex"], 0);
    glDrawArrays(GL_TRIANGLES, 0, 6);
}

