#include "hip/hip_runtime.h"
#include "raytracer.cu"

using namespace Progression;

int main(int argc, char* argv[]) {
    auto conf = PG::config::Config(PG_ROOT_DIR "configs/default.toml");
    if (!conf) {
        std::cout << "could not parse config file" << std::endl;
        exit(0);
    }

    PG::EngineInitialize(conf);

    auto scene = Scene::Load("/home/liam/Documents/School/5351/realtime-cuda-raytracer/rayTrace.pgscn");
    auto camera = scene->GetCamera();
    camera->AddComponent<UserCameraComponent>(new UserCameraComponent(camera));

    RayTracer rayTracer;
    int numSpheres = 400;
    rayTracer.Init(1, numSpheres, 5);

    Sphere h_spheres[numSpheres];
    for (int i = 0; i < numSpheres; ++i) {
        // h_spheres[i] = Sphere(make_float3(-6 + 3*i, 0, -10), 1, i);
        float X = rand() / (float) RAND_MAX * 20.f - 10;
        float Y = rand() / (float) RAND_MAX * 20.f - 10;
        float Z = rand() / (float) RAND_MAX * 20.f - 10;
        h_spheres[i] = Sphere(make_float3(X, Y, Z), 1, rand() % 5);
    }

    RTMaterial h_mats[5];
    h_mats[0].kd = make_float3(1, 0, 0);
    h_mats[1].kd = make_float3(0, 1, 0);
    h_mats[2].kd = make_float3(.4, .4, .4);
    h_mats[3].kd = make_float3(1, 1, 0);
    h_mats[4].kd = make_float3(1, 0, 1);
    for (int i = 0; i < 5; ++i) {
        h_mats[i].ks = make_float3(.7, .7, .7);
        h_mats[i].power = 50;
    }

    float3 lights[2];
    lights[0] = normalize(make_float3(0, 0, -1));
    lights[1] = make_float3(1, 1, 1);

    // check(hipMalloc((void**) &rayTracer.d_spheres, 5 * sizeof(Sphere)));
    check(hipMemcpy(rayTracer.scene.spheres, h_spheres, sizeof(Sphere) * numSpheres, hipMemcpyHostToDevice));
    rayTracer.scene.numSpheres = numSpheres;
    check(hipMemcpy(rayTracer.scene.materials, h_mats, sizeof(RTMaterial) * 5, hipMemcpyHostToDevice));
    check(hipMemcpy(rayTracer.scene.lights, lights, 1 * sizeof(float3) * 2, hipMemcpyHostToDevice));
    rayTracer.scene.numDirectionalLights = 1;
    rayTracer.scene.numPointLights = 0;

    Window::SetRelativeMouse(true);
    PG::Input::PollEvents();
    while (!PG::EngineShutdown) {
        PG::Window::StartFrame();
        PG::Input::PollEvents();

        if (PG::Input::GetKeyDown(PG::PG_K_ESC))
            PG::EngineShutdown = true;

        // scene->Update();
        camera->Update();

        rayTracer.Render(camera);

        PG::Window::EndFrame();
    }

    rayTracer.Free();

    PG::EngineQuit();

    return 0;
}
