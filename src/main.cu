#include "hip/hip_runtime.h"
#include "raytracer.cu"
#include "resource_loader.h"

using namespace Progression;

int main(int argc, char* argv[]) {
    auto conf = PG::config::Config(PG_ROOT_DIR "configs/default.toml");
    if (!conf) {
        std::cout << "could not parse config file" << std::endl;
        exit(0);
    }

    PG::EngineInitialize(conf);

	auto scene = Scene::Load(PG_ROOT_DIR "../../rayTrace.pgscn");
	//auto scene = Scene::Load("/home/liam/Documents/School/5351/realtime-cuda-raytracer/rayTrace.pgscn");
	auto camera = scene->GetCamera();
    camera->AddComponent<UserCameraComponent>(new UserCameraComponent(camera));

    RayTracer rayTracer;
    const int numSpheres = 400;
    rayTracer.Init(1, numSpheres, 5);

    // spheres
    Sphere h_spheres[numSpheres];
    for (int i = 0; i < numSpheres; ++i) {
        // h_spheres[i] = Sphere(make_float3(-6 + 3*i, 0, -10), 1, i);
        float X = rand() / (float) RAND_MAX * 20.f - 10;
        float Y = rand() / (float) RAND_MAX * 20.f - 10;
        float Z = rand() / (float) RAND_MAX * 20.f - 10;
        h_spheres[i] = Sphere(make_float3(X, Y, Z), 1, rand() % 5);
    }

    check(hipMemcpy(rayTracer.scene.spheres, h_spheres, sizeof(Sphere) * numSpheres, hipMemcpyHostToDevice));
    rayTracer.scene.numSpheres = numSpheres;
    rayTracer.scene.numSpheres = 0;


    /*
    std::vector<glm::vec3> verts = {
        glm::vec3(-5, 5, 0),
        glm::vec3(-5, -5, 0),
        glm::vec3(5, -5, 0)
    };
    std::vector<glm::vec3> norms = {
        glm::vec3(0, 0, 1),
        glm::vec3(0, 0, 1),
        glm::vec3(0, 0, 1)
    };
    std::vector<Triangle> tris = {
        Triangle(0, 1, 2)
    };
    rayTracer.scene.mesh = CudaMesh(verts, norms, tris, 0);
    */
    const auto& mesh = loadRTModel(PG_ROOT_DIR "../../cube.rtModel");
    rayTracer.scene.mesh = mesh[0].first;


    // materials
    RTMaterial h_mats[5];
    h_mats[0].kd = make_float3(1, 0, 0);
    h_mats[1].kd = make_float3(0, 1, 0);
    h_mats[2].kd = make_float3(.4, .4, .4);
    h_mats[3].kd = make_float3(1, 1, 0);
    h_mats[4].kd = make_float3(1, 0, 1);
    for (int i = 0; i < 5; ++i) {
        h_mats[i].ks = make_float3(.7, .7, .7);
        h_mats[i].power = 50;
    }

    check(hipMemcpy(rayTracer.scene.materials, h_mats, sizeof(RTMaterial) * 5, hipMemcpyHostToDevice));


    // lights
    float3 lights[2];
    lights[0] = normalize(make_float3(.8, -.3, -1));
    lights[1] = make_float3(1, 1, 1);

    check(hipMemcpy(rayTracer.scene.lights, lights, 1 * sizeof(float3) * 2, hipMemcpyHostToDevice));
    rayTracer.scene.numDirectionalLights = 1;
    rayTracer.scene.numPointLights = 0;

    Window::SetRelativeMouse(true);
    PG::Input::PollEvents();
    while (!PG::EngineShutdown) {
        PG::Window::StartFrame();
        PG::Input::PollEvents();

        if (PG::Input::GetKeyDown(PG::PG_K_ESC))
            PG::EngineShutdown = true;

        // scene->Update();
        camera->Update();

        rayTracer.Render(camera);

        PG::Window::EndFrame();
    }

    rayTracer.Free();

    PG::EngineQuit();

    return 0;
}
